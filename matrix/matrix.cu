
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__
void add(double *a, double *b, double *c, long size){
  long stride = blockDim.x;
  long index = blockIdx.x*stride+threadIdx.x;

   for (long i=index; i < size; i+=stride)
     for (long j=0; j < size; j++){
       c[i*size+j] = 0.0;
       for (long k=0; k < size; k++)
         c[i*size+j] += a[i*size+k] * b[k*size+j];
    }
}

void Test(void);

int main(int argc, char *argv[])
{
    printf("testing... \n");
    long size = argc==1 ? 3 : atol(argv[1]);

  double A[size][size];
  double B[size][size];
  double C[size][size];

  A[0][0] = 14.0; A[0][1] = 9.0;  A[0][2] = 3.0;
  A[1][0] = 2.0;  A[1][1] = 11.0; A[1][2] = 15.0;
  A[2][0] = 0.0;  A[2][1] = 12.0; A[2][2] = 17.0;

  B[0][0] = 12.0; B[0][1] = 25.0; B[0][2] = 5.0;
  B[1][0] = 9.0;  B[1][1] = 10.0; B[1][2] = 0.0;
  B[2][0] = 8.0;  B[2][1] = 5.0;  B[2][2] = 1.0;

//  printf("test matrix A... \n");
//    for(int i=0;i<size;i++){
//        for(int j=0;j<size;j++)
//            printf("%4.1f ",A[i][j]);
//        printf("\n");
//    }
//
//  printf("test matrix B... \n");
//    for(int i=0;i<size;i++){
//        for(int j=0;j<size;j++)
//            printf("%4.1f ",B[i][j]);
//        printf("\n");
//    }

    printf("actual work ... \n");
    double *a, *b,*c;

    hipMallocManaged(&a,sizeof(double)*size*size);
    hipMallocManaged(&b,sizeof(double)*size*size);
    hipMallocManaged(&c,sizeof(double)*size*size);


   for (long i=0; i < 3; i++)
     for (long j=0; j < 3; j++){
       a[i*3+j] = A[i][j];
       b[i*3+j] = B[i][j];
     }

    add<<<2,1024>>>(a,b,c,size);

    hipDeviceSynchronize();

//    for (long i=0; i < size; i++){
//        for (long j=0; j < size; j++)
//            printf("%4.1f ", c[i*size+j]);
//        printf("\n");
//    }

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
